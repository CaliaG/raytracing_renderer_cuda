#include "hip/hip_runtime.h"
﻿#include "common.h"
#include "vec3.h"
#include "ray.h"
#include "sphere.h"
#include "hitable_list.h"
#include "camera.h"
#include "hiprand/hiprand_kernel.h"

// remember, the # converts the definition to a char*
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__)
//#define constructo

inline void check_cuda(hipError_t errcode, char const* const func, const char* const file, int const line) {
    if (errcode) {
        fprintf(stderr, "check_cuda error (%d):\nFile \"%s\", line %d\n%s\n",
            static_cast<unsigned int>(errcode), file, line, hipGetErrorString(errcode));
        hipDeviceReset();
        exit(99);
    }
}

#define WIDTH 1200
#define HEIGHT 600

#define SAMPLES_PER_PIXEL 100

#define SEED 1000

// we will divide the work on the GPU into blocks of 8x8 threads beacause
// 1 - can be multiplied to 32 so they can fit into warps easily
// 2 - is small so it helps similar pixels do similar work
#define THREAD_SIZE_X 8
#define THREAD_SIZE_Y 8

// just to make things easier
__host__ __device__ constexpr int XY(int x, int y) {
    // change to intrinsic
    return y * WIDTH + x;
}

__device__ vec3 color(const ray& r, hitable_list** scene) {
    hit_record hrec;
    if ((*scene)->hit(r, 0.f, FLT_MAX, hrec)) {
        return 0.5f * vec3(hrec.n().x() + 1.0f, hrec.n().y() + 1.0f, hrec.n().z() + 1.0f);
    } else {
        vec3 unit_direction = vec3::normalize(r.direction());
        float t = 0.5f * (unit_direction.y() + 1.0f);
        return (1.0f - t) * vec3(1.0, 1.0, 1.0) + t * vec3(0.5, 0.7, 1.0);
    }
}

__global__ void init_rand_state(hiprandState* randState, int width, int height) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    // if out of range
    if ((i >= width) || (j >= height)) {
        return;
    }

    int index = XY(i, j);
    
    // same seed for every thread
    hiprand_init(SEED, index, 0, &randState[index]);
}

__global__ void render(vec3* frameBuffer, int width, int height,
    hitable_list** scene,
    camera** cam,
    hiprandState* randState) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    // if out of range
    if ((i >= width) || (j >= height)) {
        return;
    }

    int index = XY(i, j);

    hiprandState rstate = randState[index];
    vec3 col;

    for (uint16_t sample = 0; sample < SAMPLES_PER_PIXEL; ++sample) {
        float u = float(i + hiprand_uniform(&rstate)) / float(width);
        float v = float(j + hiprand_uniform(&rstate)) / float(height);
        ray r = (*cam)->get_ray(u, v);
        col += color(r, scene);
    }

    frameBuffer[index] = col / float(SAMPLES_PER_PIXEL);
}

// TODO: check for array boundary
__global__ void populate_scene(hitable_object** objects, hitable_list** scene, camera** cam) {
    if (threadIdx.x == 0 && blockIdx.x == 0) { // only call once
        *(objects) = new sphere(vec3(0, 0, -1), 0.5);
        *(objects + 1) = new sphere(vec3(0, -100.5, -1), 100);
        *scene = new hitable_list(objects, 2);
        *cam = new camera();
    }
}

__global__ void free_scene(hitable_object** objects, hitable_list** scene, camera** cam) {
    // Objects already destoryed inside scene
    //delete* (objects);
    //delete* (objects + 1);
    delete* scene;
    delete* cam;
}

int main() {

    std::cout << "Rendering a " << WIDTH << "x" << HEIGHT << " image ";
    std::cout << "(" << SAMPLES_PER_PIXEL << " samples per pixel) ";
    std::cout << "in " << THREAD_SIZE_X << "x" << THREAD_SIZE_Y << " blocks.\n";

    // _d stands for device
    hitable_object** hitableObjects_d;
    hitable_list** scene_d;
    camera** camera_d;

    // random state
    hiprandState* rand_state_d;
    checkCudaErrors(hipMalloc((void**)&rand_state_d, WIDTH * HEIGHT * sizeof(hiprandState)));

    // allocate unified memory that holds the size of our image
    vec3* frameBuffer_u; // u stands for unified
    size_t frameBufferSize = WIDTH * HEIGHT * sizeof(vec3); // RGB values for each pixel
    checkCudaErrors(hipMallocManaged((void**)&frameBuffer_u, frameBufferSize));

    // allocate device memory
    checkCudaErrors(hipMalloc((void**)&hitableObjects_d, 2 * sizeof(hitable_object*)));
    checkCudaErrors(hipMalloc((void**)&scene_d, sizeof(hitable_list*)));
    checkCudaErrors(hipMalloc((void**)&camera_d, sizeof(camera*)));

    // remember, construction is done in 1 block, 1 thread
    populate_scene<<<1, 1>>> (hitableObjects_d, scene_d, camera_d);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    auto start = std::chrono::system_clock::now();
    
    // remember: always round with + 1
    dim3 blocks(WIDTH / THREAD_SIZE_X + 1, HEIGHT / THREAD_SIZE_Y + 1);
    dim3 threads(THREAD_SIZE_X, THREAD_SIZE_Y);

    // init rand state for each pixel
    init_rand_state<<<blocks,threads>>>(rand_state_d, WIDTH, HEIGHT);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    render<<<blocks, threads>>>(frameBuffer_u, WIDTH, HEIGHT,
        scene_d,
        camera_d,
        rand_state_d);

    checkCudaErrors(hipGetLastError());
    // block host until all device threads finish
    checkCudaErrors(hipDeviceSynchronize());

    auto end = std::chrono::system_clock::now();

    auto timer_seconds = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
    std::cout <<  "took " << timer_seconds << "us.\n";

    // Output frame buffer as a ppm image
    std::ofstream ppm_image("render.ppm");
    ppm_image << "P3\n" << WIDTH << " " << HEIGHT << "\n255\n";
    for (int j = HEIGHT - 1; j >= 0; j--) {
        for (int i = 0; i < WIDTH; i++) {
            size_t index = XY(i, j);
            float r = frameBuffer_u[index].r();
            float g = frameBuffer_u[index].g();
            float b = frameBuffer_u[index].b();
            int ir = int(255.99 * r);
            int ig = int(255.99 * g);
            int ib = int(255.99 * b);
            ppm_image << ir << " " << ig << " " << ib << "\n";
        }
    }
    ppm_image.close();

    // clean everything
    checkCudaErrors(hipDeviceSynchronize());
    free_scene<<<1, 1>>>(hitableObjects_d, scene_d, camera_d);
    checkCudaErrors(hipGetLastError());

    checkCudaErrors(hipFree(hitableObjects_d));
    checkCudaErrors(hipFree(scene_d));
    checkCudaErrors(hipFree(camera_d));
    checkCudaErrors(hipFree(rand_state_d));
    checkCudaErrors(hipFree(frameBuffer_u));

    // Documentation: Destroy all allocations and reset all state on the
    // current device in the current process
    checkCudaErrors(hipDeviceReset());

    return 0;
}