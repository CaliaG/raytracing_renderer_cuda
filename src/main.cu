#include "hip/hip_runtime.h"
﻿#include "common.h"
#include "vec3.h"
#include "ray.h"
#include "sphere.h"
#include "hitable_list.h"
#include "camera.h"
#include "bvh.h"
#include "texture.h"

#define STB_IMAGE_IMPLEMENTATION
#include "libs/stb/stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "libs/stb/stb_image_write.h"

#define SAMPLES_PER_PIXEL 100

#define SCENE_BALLS
//#define SCENE_HDR

void save_to_jpg(vec3* frameBuffer_u);
void save_to_ppm(vec3* frameBuffer_u);

// remember, the # converts the definition to a char*
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__)

inline void check_cuda(hipError_t errcode, char const* const func, const char* const file, int const line) {
    if (errcode) {
        fprintf(stderr, "check_cuda error (%d):\nFile \"%s\", line %d\n%s\n",
            static_cast<unsigned int>(errcode), file, line, hipGetErrorString(errcode));
        hipDeviceReset();
        exit(99);
    }
}

//texture<float, 2, hipReadModeElementType> tex;

__device__ vec3 color(const ray& r, hitable_list** scene, hiprandState* rstate) {
    // this section is a simple implementation for a diffuse material with a 50%
    // attenuation at each bounce
    ray curr_r = r;
    vec3 curr_attenuation(1.f, .8f, .7f);
    //vec3 curr_attenuation(0.067, 0.471, 0.576);
    for (int i = 0; i < RAY_BOUNCES; ++i) {
        hit_record hrec;
        // 0.001 -> ignore hits near zero
        if ((*scene)->hit(curr_r, 0.00001f, FLT_MAX, hrec)) {
            ray scattered;
            vec3 attenuation;

            vec3 emit = hrec.m()->emit(hrec) + vec3(0.1,0.1,0.1); // bloomy effect
            if (hrec.m()->scatter(curr_r, scattered, hrec, attenuation, rstate)) {
                curr_attenuation = emit + attenuation*curr_attenuation;
                curr_r = scattered;
            } else {
                return emit;
            }

            /*vec3 target = hrec.p() + hrec.n() + random_point_unit_sphere(rstate);
            curr_attenuation *= 0.5f;
            curr_r = ray(hrec.p(), target - hrec.p());*/
        } else {
            /*vec3 unit_direction = vec3::normalize(curr_r.direction());
            float t = 0.5f * (unit_direction.y() + 1.0f);
            vec3 v = (1.0f - t) * vec3(1.0, 1.0, 1.0) + t * vec3(0.5, 0.7, 1.0);
            return curr_attenuation * v;
            */
            // return world color
            return curr_attenuation;
        }
    }
    return vec3(); // exceeded recursion
    /*if ((col.r() < 0) || (col.g() < 0) || (col.b() < 0)) {
    printf("ERROR: COL=%f,%f,%f\n", col.r(), col.g(), col.b());
    }*/
}

__global__ void init_rand_state(hiprandState* randState, int width, int height) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    // -- if out of range
    if ((i >= width) || (j >= height)) {
        return;
    }

    int index = utils::XY(i, j);
    
    // -- same seed for every thread, very slow
    //hiprand_init(SEED, index, 0, &randState[index]);

    // -- different seed for each thread, fast
    hiprand_init(SEED + index, 0, 0, &randState[index]);

    // -- produces weird artifacts
    //hiprand_init(SEED, 0, 0, &randState[index]);
}

__global__ void render(vec3* frameBuffer, int width, int height,
    hitable_list** scene,
    camera** cam,
    hiprandState* randState) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    // -- if out of range
    if ((i >= width) || (j >= height)) {
        return;
    }

    int index = utils::XY(i, j);

    hiprandState rstate = randState[index];
    vec3 col;

    for (uint16_t sample = 0; sample < SAMPLES_PER_PIXEL; ++sample) {
        // -- remember: random value is [0, 1]
        float u = float(i + hiprand_uniform(&rstate)) / float(width);
        float v = float(j + hiprand_uniform(&rstate)) / float(height);
        ray r = (*cam)->get_ray(u, v, &rstate);
        col += color(r, scene, &rstate);
        
    }

    col /= float(SAMPLES_PER_PIXEL);
    //col.saturate();
    // -- do gamma correction with gamma 2 => raise the color to the power of 1/2 (sqrt)
    frameBuffer[index] = col.saturate().gamma_correct();

    // -- only for debug
    //frameBuffer[index] = col.gamma_correct();
}

#ifdef SCENE_HDR
constexpr char imagePath[] = "textures/hdr.jpg";
__global__ void populate_scene_hdr(hitable_object** objects, hitable_list** scene, 
                                    camera** cam, hiprandState* state, float* textureBuffer
) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        objects[0] = new sphere(
            vec3(1., 0, -1),
            1,
            //new lambertian(new constant_texture(vec3(0.6, 0.1, 0.1)))
            new metal(vec3(0.8, 0.2, 0.5), 0.05)
        );
        objects[0]->set_id(0);

        text* hdr_texture = new image_texture(textureBuffer, WIDTH*2, HEIGHT*2);
        //sphere 2
        objects[1] = new sphere(
            vec3(0, 0, 0),
            10,
            new emitter(hdr_texture)
        );
        objects[1]->set_id(1);

        objects[2] = new sphere(
            vec3(-1., 0, -1),
            1,
            new lambertian(new constant_texture(vec3(0.6, 0.1, 0.1)))
        );
        objects[2]->set_id(2);

        *scene = new hitable_list(objects, nullptr, 3);
        scene[0]->set_id(3);

        vec3 lookfrom = vec3(-1, 2, 9);
        vec3 lookat = vec3(0, 0, -1);
        float dist_to_focus = (lookfrom - lookat).length();
        float aperture = .25f;
        *cam = new camera(
            lookfrom, // lookfrom
            lookat, // lookat
            vec3(0, 1, 0),   // up
            20.f,           // fov
            float(WIDTH) / float(HEIGHT),
            aperture,
            dist_to_focus,
            0,
            0.2
        );
    }
}
#endif

// TODO: check for array boundary
#ifdef SCENE_BALLS
constexpr char imagePath[] = "textures/earth.jpg";
__global__ void populate_scene_balls(hitable_object** objects, hitable_list** scene,
                                      camera** cam, hiprandState* state, float* textureBuffer
) {
    if (threadIdx.x == 0 && blockIdx.x == 0) { // only call once
        // sphere 1
        objects[0] = new sphere(
            vec3(0, 0, -1),
            0.5,
            new lambertian(new constant_texture(vec3(0.6, 0.1, 0.1)))
            //new dielectric(1.3, vec3(1, 1, 1))
            //new dielectric(1.5, vec3(1,1,1))
        );
        objects[0]->set_id(0);
        
        // -- sphere 2
        /*text* checker = new checker_texture(
            new constant_texture(vec3(0.1, 0.2, 0.5)),
            new constant_texture(vec3(0.5, 0.2, 0.1)));*/
        //text* noise1 = new noise_texture(noise_type::TURBULANCE, .1f);
        text* noise1 = new noise_texture(noise_type::MARBLE, 1.f);
        /*text* noise = new wood_texture(vec3(0.792, 0.643, 0.447),
            //vec3(0.267, 0.188, 0.133),
            vec3(0.412, 0.349, 0.306),
            10.f);*/
        /*text* checker = new checker_texture(
            noise,
            noise);*/

        //text* noise1 = new noise_texture(noise_type::MARBLE, .2f);
        objects[1] = new sphere(
            vec3(0, -1000.5, 1),
            1000,
            //10,
            new lambertian(noise1)
            //new lambertian(new constant_texture(vec3(0.1, 0.2, 0.5)))
        );
        objects[1]->set_id(1);
        /*objects[1] = new sphere(
            vec3(0, -20, 1),
            10,
            new lambertian(vec3(0.1, 0.2, 0.5))
        );
        objects[1]->set_id(1);*/

        text* im_text = new image_texture(textureBuffer, 1200, 600);
        // -- sphere 3
        objects[2] = new sphere(
            vec3(1, 0, -1),
            0.5,
            //new dielectric(1.5)
            //new lambertian(noise1)
            //new lambertian(new constant_texture(vec3(0.1, 0.2, 0.5)))
            new emitter(im_text,2),
            //new metal(vec3(1.f), 0.f)
            //new metal(vec3(1.f), 0.f)
            //new metal(vec3(0.075, 0.461, 0.559), 0.1f)
        true);
        objects[2]->set_id(2);

        // -- sphere 4
        //perlin_noise::init(state);
        //perlin_noise noise;
        //text* per_text = new noise_texture(state);

        objects[3] = new sphere( vec3(-1, 0, -2), 0.5,
            //new lambertian(per_text)
            new metal(vec3(1.f), 0.f)
            //new lambertian(new constant_texture(vec3(0.6, 0.1, 0.1)))
            //new dielectric(1.5, vec3(1, 1, 1))
            //new metal(vec3(0.8, 0.8, 0.8), 0.5)
        );
        objects[3]->set_id(3);

        // -- sphere 5
        objects[4] = new sphere(vec3(0, 0, -2), 0.5, new metal(vec3(0.8, 0.8, 0.8), 0.5));
        objects[4]->set_id(4);
        
        objects[5] = new sphere(
            vec3(1, 0, -2),
            0.5,
            //new emitter(vec3(1,0.5,0.5))

            new dielectric(1.5, vec3(1, 1, 1))
            //new lambertian(new constant_texture(vec3(0.1, 0.2, 0.5)))
            //new lambertian(vec3(0.2, 0.9, 0.3)*0.6)
            
        );
        objects[5]->set_id(5);

        
        objects[6]= new sphere(
            vec3(-1, 0, -1),
            0.5,
            new emitter(new constant_texture(vec3(0.5,1,0.5)))
            //new dielectric(1.1, vec3(0.8,1.0,0.8))
        );
        objects[6]->set_id(6);

        objects[7] = new moving_sphere(
            vec3(-1, 1, -1),
            vec3(-2, 1, -1),
            0.f,
            1.f,
            0.2,

            new lambertian(new constant_texture(vec3(0.6, 0.1, 0.1)))
            //new dielectric(1.5, vec3(1, 1, 1))
            //new metal(vec3(0.8, 0.8, 0.8), 0.5)
        );
        /*objects[7] = new sphere(
            vec3(-1, 1, -1),
            0.5,
            new lambertian(vec3(0.6, 0.1, 0.1)));*/
        objects[7]->set_id(7);

        objects[8] = new bvh_node(objects, 8, 0, 1, state, 0);
        objects[8]->set_id(8);

        // check bvh hierarchy
        //bvh_node::display_tree(static_cast<bvh_node*>(objects[8]), 2);

        *scene = new hitable_list(objects, static_cast<bvh_node*>(objects[8]), 8);
        scene[0]->set_id(9);

        //for (int i = 0; i < 9; ++i) {
        //    printf("(%d) %s\n", objects[i]->get_id(), hitable_object::obj_type_str(objects[i]->get_object_type()));
        //}

        //vec3 lookfrom = vec3(-2, 1, 2) * 2;
        vec3 lookfrom = vec3(-1, 1,5); // revert to 2
        //THISvec3 lookfrom = vec3(5, 2, 3);
        //vec3 lookat = vec3(0, 0, -1);
        //vec3 lookat = vec3(-1, 0, -1); // redball
        //vec3 lookat = vec3(1, 0, -1); // marble ball
        vec3 lookat = vec3(0, 0, -1);
        float dist_to_focus = (lookfrom - lookat).length();
        float aperture = .25f;
        *cam = new camera(
            lookfrom, // lookfrom
            lookat, // lookat
            vec3(0,1,0),   // up
            20.f,           // fov
            float(WIDTH) / float(HEIGHT),
            aperture,
            dist_to_focus,
            0,
            0.2
        );

        //hit_record hrec;
        //ray r = (*cam)->get_ray(0.54, 0.5, state);
        //static_cast<bvh_node*>(objects[8])->dfs(r, 0.001f, FLT_MAX, hrec);
        //assert(0);
    }
}
#endif

__global__ void free_scene(hitable_object** objects, hitable_list** scene, camera** cam) {
    // Objects already destoryed inside scene
    //delete* (objects);
    //delete* (objects + 1);
    delete* scene;
    delete* cam;
}

int main(int argc, char** argv) {
    // loading image to host
    
    // load image as uint8_t
    //uint8_t* imgData = stbi_load(imagePath, &w, &h, &ch, 0);
    //stbi_write_png("export.png", w, h, ch, imgData, w * ch);

    // load image as float
    int w, h, ch;
    stbi_ldr_to_hdr_scale(1.0f);
    stbi_ldr_to_hdr_gamma(1.0f);
    float* imgData_h = stbi_loadf(imagePath, &w, &h, &ch, 0);
    std::cout << "Loaded image with " << w << "x" << h << " and " << ch << " channels\n";

    float* imgData_d;
    size_t imgSize = w * h * ch * sizeof(float);
    // TODO: for now, store texture in global memory. In the future, use texture
    checkCudaErrors(hipMalloc((float**)&imgData_d, imgSize));
    checkCudaErrors(hipMemcpy(imgData_d, imgData_h, imgSize, hipMemcpyHostToDevice));
    stbi_image_free(imgData_h);
   
    //stbi_write_png("export2.png", w, h, ch, imgData, w * ch);
    //stbi_image_free(imgData);

    std::cout << "Rendering a " << WIDTH << "x" << HEIGHT << " image ";
    std::cout << "(" << SAMPLES_PER_PIXEL << " samples per pixel) ";
    std::cout << "in " << THREAD_SIZE_X << "x" << THREAD_SIZE_Y << " blocks.\n";

    // _d stands for device
    hitable_object** hitableObjects_d;
    hitable_list** scene_d;
    camera** camera_d;

    // random state
    hiprandState* rand_state_d;
    checkCudaErrors(hipMalloc((void**)&rand_state_d, WIDTH * HEIGHT * sizeof(hiprandState)));

    // allocate unified memory that holds the size of our image
    vec3* frameBuffer_u; // u stands for unified
    size_t frameBufferSize = WIDTH * HEIGHT * sizeof(vec3); // RGB values for each pixel
    checkCudaErrors(hipMallocManaged((void**)&frameBuffer_u, frameBufferSize));

    // allocate device memory
#ifdef SCENE_BALLS
    checkCudaErrors(hipMalloc((void**)&hitableObjects_d, 9 * sizeof(hitable_object*)));
#endif
#ifdef SCENE_HDR
    checkCudaErrors(hipMalloc((void**)&hitableObjects_d, 3 * sizeof(hitable_object*)));
#endif
    checkCudaErrors(hipMalloc((void**)&scene_d, sizeof(hitable_list*)));
    checkCudaErrors(hipMalloc((void**)&camera_d, sizeof(camera*)));

    // remember, construction is done in 1 block, 1 thread
#ifdef SCENE_BALLS
    populate_scene_balls<<<1, 1>>>(hitableObjects_d, scene_d, camera_d, rand_state_d, imgData_d);
#endif
#ifdef SCENE_HDR
    populate_scene_hdr<<<1, 1>>>(hitableObjects_d, scene_d, camera_d, rand_state_d, imgData_d);
#endif
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    clock_t start, stop;
    start = clock();
    
    // remember: always round with + 1
    dim3 blocks(WIDTH / THREAD_SIZE_X + 1, HEIGHT / THREAD_SIZE_Y + 1);
    dim3 threads(THREAD_SIZE_X, THREAD_SIZE_Y);

    // init rand state for each pixel
    init_rand_state<<<blocks,threads>>>(rand_state_d, WIDTH, HEIGHT);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    render<<<blocks, threads>>>(frameBuffer_u, WIDTH, HEIGHT, scene_d, camera_d, rand_state_d);

    checkCudaErrors(hipGetLastError());
    // block host until all device threads finish
    checkCudaErrors(hipDeviceSynchronize());

    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cout << "took " << timer_seconds << " seconds.\n";

    // -- Output frame buffer as a jpg image
    save_to_jpg(frameBuffer_u);
    // -- Output frame buffer as a ppm image
    save_to_ppm(frameBuffer_u);

    // clean everything
    checkCudaErrors(hipDeviceSynchronize());
    free_scene<<<1, 1>>>(hitableObjects_d, scene_d, camera_d);
    checkCudaErrors(hipGetLastError());

    checkCudaErrors(hipFree(hitableObjects_d));
    checkCudaErrors(hipFree(scene_d));
    checkCudaErrors(hipFree(camera_d));
    checkCudaErrors(hipFree(rand_state_d));
    checkCudaErrors(hipFree(frameBuffer_u));

    checkCudaErrors(hipFree(imgData_d));

    // Documentation: Destroy all allocations and reset all state on the
    // current device in the current process
    checkCudaErrors(hipDeviceReset());

    return 0;
}

void save_to_jpg(vec3* frameBuffer_u) {
    uint8_t* imgBuff = (uint8_t*)std::malloc(WIDTH * HEIGHT * 3 * sizeof(uint8_t));
    for (int j = HEIGHT - 1; j >= 0; --j) {
        for (int i = 0; i < WIDTH; ++i) {
            size_t index = utils::XY(i, j);
            // stbi generates a Y flipped image
            size_t rev_index = utils::XY(i, HEIGHT - j - 1);
            float r = frameBuffer_u[index].r();
            float g = frameBuffer_u[index].g();
            float b = frameBuffer_u[index].b();
            imgBuff[rev_index * 3 + 0] = int(255.999f * r) & 255;
            imgBuff[rev_index * 3 + 1] = int(255.999f * g) & 255;
            imgBuff[rev_index * 3 + 2] = int(255.999f * b) & 255;
        }
    }
    //stbi_write_png("out.png", WIDTH, HEIGHT, 3, imgBuff, WIDTH * 3);
    stbi_write_jpg("out.jpg", WIDTH, HEIGHT, 3, imgBuff, 100);
    std::free(imgBuff);
}

void save_to_ppm(vec3* frameBuffer_u) {
    std::ofstream ppm_image("out.ppm");
    ppm_image << "P3\n" << WIDTH << " " << HEIGHT << "\n255\n";
    for (int j = HEIGHT - 1; j >= 0; j--) {
        for (int i = 0; i < WIDTH; i++) {
            size_t index = utils::XY(i, j);
            float r = frameBuffer_u[index].r();
            float g = frameBuffer_u[index].g();
            float b = frameBuffer_u[index].b();
            int ir = int(255.99 * r);
            int ig = int(255.99 * g);
            int ib = int(255.99 * b);
            ppm_image << ir << " " << ig << " " << ib << "\n";
        }
    }
    ppm_image.close();
}