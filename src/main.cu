#include "hip/hip_runtime.h"
﻿#include "common.h"
#include "vec3.h"
#include "ray.h"
#include "sphere.h"
#include "hitable_list.h"
#include "camera.h"
#include "hiprand/hiprand_kernel.h"

// remember, the # converts the definition to a char*
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__)
//#define constructo

inline void check_cuda(hipError_t errcode, char const* const func, const char* const file, int const line) {
    if (errcode) {
        fprintf(stderr, "check_cuda error (%d):\nFile \"%s\", line %d\n%s\n",
            static_cast<unsigned int>(errcode), file, line, hipGetErrorString(errcode));
        hipDeviceReset();
        exit(99);
    }
}

#define WIDTH 1200
#define HEIGHT 600

#define SAMPLES_PER_PIXEL 100

#define SEED 1000

// we will divide the work on the GPU into blocks of 8x8 threads beacause
// 1 - can be multiplied to 32 so they can fit into warps easily
// 2 - is small so it helps similar pixels do similar work
#define THREAD_SIZE_X 8
#define THREAD_SIZE_Y 8

// just to make things easier
__host__ __device__ constexpr int XY(int x, int y) {
    // change to intrinsic
    return y * WIDTH + x;
}

__device__ vec3 random_point_unit_sphere(hiprandState* rstate) {
    vec3 point;
    do {
        // grab a random point and center it in
        // the unit circle
        // the random value is generated using the random state
        // of the pixel calling the function
        point = 2.f * vec3(
            hiprand_uniform(rstate),
            hiprand_uniform(rstate),
            hiprand_uniform(rstate)
        ) - vec3(1.f, 1.f, 1.f);

    } while (point.sq_length() >= 1.f);
    return point;
}

__device__ vec3 color(const ray& r, hitable_list** scene, hiprandState* rstate) {

    // this section is a simple implementation for a diffuse material with a 50%
    // attenuation at each bounce
    ray curr_r = r;
    float curr_attenuation = 1.f;
    int bounces = 50;
    for (int i = 0; i < bounces; ++i) {
        hit_record hrec;
        // 0.001 -> ignore hits near zero
        if ((*scene)->hit(curr_r, 0.001f, FLT_MAX, hrec)) {
            vec3 target = hrec.p() + hrec.n() + random_point_unit_sphere(rstate);
            curr_attenuation *= 0.5f;
            curr_r = ray(hrec.p(), target - hrec.p());
        } else {
            vec3 unit_direction = vec3::normalize(curr_r.direction());
            float t = 0.5f * (unit_direction.y() + 1.0f);
            vec3 v = (1.0f - t) * vec3(1.0, 1.0, 1.0) + t * vec3(0.5, 0.7, 1.0);
            return curr_attenuation * v;
        }
    }
    return vec3(); // exceeded recursion
}

__global__ void init_rand_state(hiprandState* randState, int width, int height) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    // if out of range
    if ((i >= width) || (j >= height)) {
        return;
    }

    int index = XY(i, j);
    
    // same seed for every thread, very slow
    //hiprand_init(SEED, index, 0, &randState[index]);

    // different seed for each thread, fast
    hiprand_init(SEED + index, 0, 0, &randState[index]);

    // produces weird artifacts
    //hiprand_init(SEED, 0, 0, &randState[index]);
}

__global__ void render(vec3* frameBuffer, int width, int height,
    hitable_list** scene,
    camera** cam,
    hiprandState* randState) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    // if out of range
    if ((i >= width) || (j >= height)) {
        return;
    }

    int index = XY(i, j);

    hiprandState rstate = randState[index];
    vec3 col;

    for (uint16_t sample = 0; sample < SAMPLES_PER_PIXEL; ++sample) {
        // remember: random value is [0, 1[ 
        float u = float(i + hiprand_uniform(&rstate)) / float(width);
        float v = float(j + hiprand_uniform(&rstate)) / float(height);
        ray r = (*cam)->get_ray(u, v);
        col += color(r, scene, &rstate);
    }
    col /= float(SAMPLES_PER_PIXEL);
    // do gamma correction with gamma 2 => raise the color to the power of
    // 1/gamma -> sqrt
    frameBuffer[index] = col.gamma_correct();
}

// TODO: check for array boundary
__global__ void populate_scene(hitable_object** objects, hitable_list** scene, camera** cam) {
    if (threadIdx.x == 0 && blockIdx.x == 0) { // only call once
        *(objects) = new sphere(vec3(0, 0, -1), 0.5);
        *(objects + 1) = new sphere(vec3(0, -100.5, -1), 100);
        *scene = new hitable_list(objects, 2);
        *cam = new camera();
    }
}

__global__ void free_scene(hitable_object** objects, hitable_list** scene, camera** cam) {
    // Objects already destoryed inside scene
    //delete* (objects);
    //delete* (objects + 1);
    delete* scene;
    delete* cam;
}

int main() {

    std::cout << "Rendering a " << WIDTH << "x" << HEIGHT << " image ";
    std::cout << "(" << SAMPLES_PER_PIXEL << " samples per pixel) ";
    std::cout << "in " << THREAD_SIZE_X << "x" << THREAD_SIZE_Y << " blocks.\n";

    // _d stands for device
    hitable_object** hitableObjects_d;
    hitable_list** scene_d;
    camera** camera_d;

    // random state
    hiprandState* rand_state_d;
    checkCudaErrors(hipMalloc((void**)&rand_state_d, WIDTH * HEIGHT * sizeof(hiprandState)));

    // allocate unified memory that holds the size of our image
    vec3* frameBuffer_u; // u stands for unified
    size_t frameBufferSize = WIDTH * HEIGHT * sizeof(vec3); // RGB values for each pixel
    checkCudaErrors(hipMallocManaged((void**)&frameBuffer_u, frameBufferSize));

    // allocate device memory
    checkCudaErrors(hipMalloc((void**)&hitableObjects_d, 2 * sizeof(hitable_object*)));
    checkCudaErrors(hipMalloc((void**)&scene_d, sizeof(hitable_list*)));
    checkCudaErrors(hipMalloc((void**)&camera_d, sizeof(camera*)));

    // remember, construction is done in 1 block, 1 thread
    populate_scene<<<1, 1>>> (hitableObjects_d, scene_d, camera_d);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    auto start = std::chrono::system_clock::now();
    
    // remember: always round with + 1
    dim3 blocks(WIDTH / THREAD_SIZE_X + 1, HEIGHT / THREAD_SIZE_Y + 1);
    dim3 threads(THREAD_SIZE_X, THREAD_SIZE_Y);

    // init rand state for each pixel
    init_rand_state<<<blocks,threads>>>(rand_state_d, WIDTH, HEIGHT);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    render<<<blocks, threads>>>(frameBuffer_u, WIDTH, HEIGHT,
        scene_d,
        camera_d,
        rand_state_d);

    checkCudaErrors(hipGetLastError());
    // block host until all device threads finish
    checkCudaErrors(hipDeviceSynchronize());

    auto end = std::chrono::system_clock::now();

    auto timer_seconds = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
    std::cout <<  "took " << timer_seconds << "us.\n";

    // Output frame buffer as a ppm image
    std::ofstream ppm_image("render.ppm");
    ppm_image << "P3\n" << WIDTH << " " << HEIGHT << "\n255\n";
    for (int j = HEIGHT - 1; j >= 0; j--) {
        for (int i = 0; i < WIDTH; i++) {
            size_t index = XY(i, j);
            float r = frameBuffer_u[index].r();
            float g = frameBuffer_u[index].g();
            float b = frameBuffer_u[index].b();
            int ir = int(255.99 * r);
            int ig = int(255.99 * g);
            int ib = int(255.99 * b);
            ppm_image << ir << " " << ig << " " << ib << "\n";
        }
    }
    ppm_image.close();

    // clean everything
    checkCudaErrors(hipDeviceSynchronize());
    free_scene<<<1, 1>>>(hitableObjects_d, scene_d, camera_d);
    checkCudaErrors(hipGetLastError());

    checkCudaErrors(hipFree(hitableObjects_d));
    checkCudaErrors(hipFree(scene_d));
    checkCudaErrors(hipFree(camera_d));
    checkCudaErrors(hipFree(rand_state_d));
    checkCudaErrors(hipFree(frameBuffer_u));

    // Documentation: Destroy all allocations and reset all state on the
    // current device in the current process
    checkCudaErrors(hipDeviceReset());

    return 0;
}