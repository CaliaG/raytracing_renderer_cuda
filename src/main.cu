#include "hip/hip_runtime.h"
﻿#include "common.h"
#include "vec3.h"
#include "ray.h"
#include "sphere.h"
#include "hitable_list.h"

// checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
// remember, the # converts the definition to a char*
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__)

void check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
    if (result) {
        std::cerr << "[CUDA ERROR " << static_cast<unsigned int>(result) << "] : " <<
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

#define WIDTH 1200
#define HEIGHT 600


// just to make things easier
__host__ __device__ constexpr int XY(int x, int y) {
    return y * WIDTH + x;
}

__device__ float hit_sphere(const vec3& center, float radius, const ray& r) {
    vec3 oc = r.origin() - center; // A - C
    // 1 - dot((p(​ t) - c)​ ,(p(​ t) - c​)) = R*R
    // 2 - dot((A​ + t*B ​- C)​ ,(A​ + t*B​ - C​)) = R*R (A is origin, B is direction)
    // 3 - t*t*dot(B,​ B)​ + 2*t*dot(B,A​-C​) + dot(A-C,A​-C​) - R*R = 0
    // we solve it as a 2nd degree polynomial with delta = b^2 - 4*a*c
    float a = vec3::dot(r.direction(), r.direction());
    float b = 2.f * vec3::dot(oc, r.direction());
    float c = vec3::dot(oc, oc) - radius * radius;
    float delta = b * b - 4 * a * c;
    if (delta < 0) {
        return -1.f;
    } else {
        return ((-b - __fsqrt_rz(delta)) / (2.f * a));
    }
}

__device__ vec3 color(const ray& r, hitable_object** world) {
    hit_record hrec;
    if ((*world)->hit(r, 0.f, FLT_MAX, hrec)) {
        return 0.5f * vec3(hrec.n().x() + 1.0f, hrec.n().y() + 1.0f, hrec.n().z() + 1.0f);
    } else {
        vec3 unit_direction = vec3::normalize(r.direction());
        float t = 0.5f * (unit_direction.y() + 1.0f);
        return (1.0f - t) * vec3(1.0, 1.0, 1.0) + t * vec3(0.5, 0.7, 1.0);
    }

    /*float h1 = hit_sphere(vec3(0.5f, -0.f, -0.8f), 0.3, r);
    if (h1 > 0.f) {
        vec3 normal = vec3::normalize(r.point_at_parameter(h1) - vec3(0.5f, -0.f, -0.8f));
        return 0.5f * vec3(normal.x() + 1.f, normal.y() + 1.f, normal.z() + 1.f);
        //return vec3(0.1f, 0.7f, 0.1f);
    }
    float h2 = hit_sphere(vec3(0.f, 0.f, -1.f), 0.5, r);
    if (h2 > 0.f) {
        vec3 normal = vec3::normalize(r.point_at_parameter(h1) - vec3(0.f, 0.f, -1.f));
        return 0.5f * vec3(normal.x() + 1.f, normal.y() + 1.f, normal.z() + 1.f);
        //return vec3(0.1f, 0.1f, 0.7f);
    }
    float h3 = hit_sphere(vec3(-0.5f, -0.f, -1.f), 0.3, r);
    if (h3 > 0.f) {
        vec3 normal = vec3::normalize(r.point_at_parameter(h1) - vec3(0.f, 0.f, -1.f));
        return 0.5f * vec3(normal.x() + 1.f, normal.y() + 1.f, normal.z() + 1.f);
        //return vec3(0.7f, 0.1f, 0.1f);
    }*/

    /*vec3 unit_direction = vec3::normalize(r.direction());
    float t = 0.5f * (unit_direction.y() + 1.f);
    //vec3 result = (1.f - t) * vec3(1.f, 1.f, 1.f) + t * vec3(0.5f, 0.7f, 1.f);
    vec3 result = (1.f - t) * vec3(1.f, 1.f, 1.f) + t * vec3(0.3f, 0.5f, 0.8f);*/
    // used for debug
    /*
        printf("OK: t = %f RDIR=%f,%f,%f UNITV=%f,%f,%f\n", t,
            r.direction().r(), r.direction().g(), r.direction().b(),
            unit_direction.r(), unit_direction.g(), unit_direction.b());
    }*/
    
}

// we will divide the work on the GPU into blocks of 8x8 threads beacause
// 1 - can be multiplied to 32 so they can fit into warps easily
// 2 - is small so it helps similar pixels do similar work
#define THREAD_SIZE_X 8
#define THREAD_SIZE_Y 8

__global__ void render(vec3* frameBuffer, int width, int height,
    vec3 lowerLeftCorner, vec3 horizontal, vec3 vertical, vec3 origin,
    hitable_object** world) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    // if out of range
    if ((i >= width) || (j >= height)) {
        return;
    }
    int index = XY(i, j);
    float u = float(i) / float(width);
    float v = float(j) / float(height);
    ray r(origin, lowerLeftCorner + u * horizontal + v * vertical);
    frameBuffer[index] = color(r, world);
    // for debug purposes
    /*
    if (j % 2 && i % 2) {
        frameBuffer[index] = vec3(float(i) / width, float(j) / height, float(j) / (width + height));
    } else {
        frameBuffer[index] = vec3();
    }*/
}

__global__ void create_world(hitable_object** d_list, hitable_object** d_world) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *(d_list) = new sphere(vec3(0, 0, -1), 0.5);
        *(d_list + 1) = new sphere(vec3(0, -100.5, -1), 100);
        *d_world = new hitable_list(d_list, 2);
    }
}

__global__ void _free(hitable_object** d_list, hitable_object** d_world) {
    delete* (d_list);
    delete* (d_list + 1);
    delete* d_world;
}

int main() {

    std::cerr << "Rendering a " << WIDTH << "x" << HEIGHT << " image ";
    std::cerr << "in " << THREAD_SIZE_X << "x" << THREAD_SIZE_Y << " blocks.\n";

    // RGB values for each pixel
    size_t frameBufferSize = WIDTH * HEIGHT * sizeof(vec3);

    vec3* frameBuffer;
    // allocate unified memory that holds the size of our image
    // remember, hipMallocManaged waits for void**
    checkCudaErrors(hipMallocManaged((void**)&frameBuffer, frameBufferSize));

    // allocate hitable objects in the device
    hitable_object** d_hitableObjects;
    checkCudaErrors(hipMalloc((void**)&d_hitableObjects, 2 * sizeof(hitable_object*)));
    hitable_object** d_world;
    checkCudaErrors(hipMalloc((void**)&d_world, sizeof(hitable_object*)));
    // remember, construction is done in 1 block, 1 thread
    create_world<<<1, 1>>> (d_hitableObjects, d_world);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    auto start = std::chrono::system_clock::now();
    
    // remember: always round with + 1
    dim3 blocks(WIDTH / THREAD_SIZE_X + 1, HEIGHT / THREAD_SIZE_Y + 1);
    dim3 threads(THREAD_SIZE_X, THREAD_SIZE_Y);

    vec3 loweLeftCorner(-2.f, -1.f, -1.f);
    vec3 horizontal(4.f, 0.f, 0.f);
    vec3 vertical(0.f, 2.f, 0.f);
    vec3 origin(0.f, 0.f, 0.f);
    render<<<blocks, threads>>>(frameBuffer, WIDTH, HEIGHT,
        loweLeftCorner, horizontal, vertical, origin,
        d_world);

    checkCudaErrors(hipGetLastError());
    // block host until all device threads finish
    checkCudaErrors(hipDeviceSynchronize());

    auto end = std::chrono::system_clock::now();

    auto timer_seconds = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
    std::cerr <<  "took " << timer_seconds << "us.\n";

    // Output frame buffer as a ppm image 
    std::cout << "P3\n" << WIDTH << " " << HEIGHT << "\n255\n";
    for (int j = HEIGHT - 1; j >= 0; j--) {
        for (int i = 0; i < WIDTH; i++) {
            size_t index = XY(i, j);
            float r = frameBuffer[index].r();
            float g = frameBuffer[index].g();
            float b = frameBuffer[index].b();
            int ir = int(255.99 * r);
            int ig = int(255.99 * g);
            int ib = int(255.99 * b);
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }

    // clean everything
    checkCudaErrors(hipDeviceSynchronize());
    _free<<<1, 1>>>(d_hitableObjects, d_world);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_hitableObjects));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(frameBuffer));

    hipDeviceReset();
    return 0;
}